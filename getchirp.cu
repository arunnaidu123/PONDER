/* Modified for FFTW3          BCJ 05/11/2004  */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <tgmath.h>
//#include <fftw3.h>

#define TWOPI 6.2831853071796
#define DFFAC 2.41e-10 /*DM (pc cm-3) = DFFAC*D (MHz) */

/* IHS getchirp routine - adapted by SMO APRIL 4 1999 */
/* revised mk */

void getchirp(float2 *chirp,int nfft,int sideband,double fsky,double bw,double dm,double z4c)
{
  double r,s,f;
  int i;
  double taper;
  //FILE *fp;
  //fp = fopen("chirp.dat","w");
  bw = bw*1e-6;
  printf( "Calculating chirp function.");
  fflush(stdout);
  printf(  "\nnfft: %d flag: %d fsky: %f bw: %f dm: %f \n",nfft,sideband,fsky,bw,dm);
  //s = TWOPI*dm/(DFFAC*(1.0+z4c/10000.0));
  s = TWOPI*dm/(DFFAC);
  
  for(i=0;i<nfft;i++) {
    f = i*bw/(nfft/2);
    
    if(f > bw) 
    {
      f -= bw; 
      f = bw -f;
    } 
    
    //r = ((double) sideband)*f*f*s/((fsky+sideband*f)*fsky*fsky);  
    if(i<=nfft/2) r = -1*f*f*s/((fsky+sideband*f)*fsky*fsky);  
    else r = f*f*s/((fsky+sideband*f)*fsky*fsky);
    
    //if(i%1000==0) printf("%lf \n",(fsky+sideband*f));
    if (f > 0.5*bw) taper = 1.0/sqrt(1.0 + pow((f/(0.94*bw)),80));
    else  taper = 1.0/sqrt(1.0 + pow(((bw-f)/(0.84*bw)),80));
//    taper = 1.0; 
      //taper = 1.0/sqrt(1.0 + pow((f/(0.94*bw)),80));
//    chirp[i] = (double)( cos(r) * taper / (float)(nfft) ) + I *
//              ( -1.0* (double)( sin(r) * taper / (float)(nfft) ));
    chirp[i].x = (float)( cos(r) * taper );
    chirp[i].y = ( -1.0* (float)( sin(r) * taper));
    //chirp[i].x = 0.0;
    //chirp[i].y = 0.0;
    //fprintf(fp,"%1.25f \n",chirp[i].x);
//    chirp[i].re = (float)( cos(r) * taper / (float)(nfft) ); 
//    chirp[i].im = -1.0* (float)( sin(r) * taper / (float)(nfft) );
//    chirp[i].im = sin(r) * taper / (nfft);

  }
//  chirp[0] = 0. + I * 0.;
  chirp[0].x = 0.0;
  chirp[0].y = 0.0;
  //fclose(fp);
//  chirp[0].re = 0.;
//  chirp[0].im = 0.;


}

