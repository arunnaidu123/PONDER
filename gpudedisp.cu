#include "hip/hip_runtime.h"
/*
/usr/local/cuda-5.5/bin/nvcc -I/usr/local/cuda-5.5/samples/common/inc -o simpleCUFFT gpudedisp.c simpleCUFFT.cu read_par.c getCoherentSampleSize.c getchirp.c allocateBuffers.c dedisperse.c send_stuff.c swap_bytes.c error_message.c strings_equal.c dedisperse_header.c freeBuffers.c -lfftw3 -lm -lpthread  -lcufft
*/
#include<stdio.h>
#include<stdlib.h>
#include"struct.cuh"
#include"global.cuh"

#include <pthread.h>
#include "dedisperse.h"
#include<time.h>
#include<sys/timeb.h>
#include<math.h>

//int stop;

char cfilename[200];
int *bufferVal;
float2 *chirp;
//char *dataIn;
float *dataOut;
float *tmp;
float2 *signal;
int site_id;
pthread_mutex_t file_mutex,stop_mutex;
struct runtime run_par;
struct argStruct arguments;
FILE *fpout;
float2 *gpuChirp; 
hipfftHandle planf,planb;
char *tempIn;
//float2 *t_signal,*f_signal;
int *dev_nfft;
float *temp;
pthread_mutex_t data_mutex;
pthread_t acquire_thread;
int dataVal=0;
long double mjdnow();
long int pointer1=0;

//void dedisperse_header(FILE *outptr);
//void *acquisition(void *p);


int *gpudedisp(void *pa)
{
  
  int i,check;
  FILE *fpin, *fpar;
  //char fileOut[200],parFile[200];
  //run_par.nacc = 2048*4;
  //run_par.threads=2;
  long int count=0,nread,pointer=0,totalSamples=0;
  long double tstamp=0.0,tempsamp;
  long double dump=0.0;
  float *tempOut;
  int stopCheck=0;
  int err;
  int flag=0;
  //foldVal = 0;
  
  fpout = fopen(cfilename,"wb");
  if(fpout == NULL)
  { 
    printf("cannot open the out put file ...... please check the spce in the location");
    exit(0);
  }
  
  //read_par( fpar, run_par.timeStamp);
  //getCoherentSampleSize();
  run_par.nfft = 1024*1024*128;
  //allocateBuffers();
  tempOut = (float*) malloc(sizeof(float)*run_par.nfft/(2*run_par.nacc));
  
  //pthread_t dedisperse_thread[run_par.threads];
  //pthread_t gpu_thread, write_thread;
  
 
  //run_par.nrdisp = run_par.nfft/2;
  
  count=0;
   
   
   //stop=0;
  machine_id = 3;
  telescope_id = 2;
  refdm = run_par.dm;
  nchans = run_par.nfft/2;
  fch1 = 334.5;
  foff = (-1)*(run_par.bw/nchans);
  nbeams = 1;
  ibeam =0;
  tsamp = run_par.dt*run_par.nacc;
  tstamp = (((long double)(run_par.nfft/4))*run_par.dt)/((long double)86400);
  tempsamp = mjdnow();
  tstart = ((double)tempsamp)+tstamp;
  //output = fpout;
  headerless = 0;
  nbands =1 ;
  nobits = 32;
  nifs = 1;
  dedisperse_header(fpout); 
  
  pthread_mutex_lock(&fold_mutex);
  foldVal=0;
  timeSeries = (float*) malloc(sizeof(float)*run_par.nfft/(2*run_par.nacc));
  pthread_mutex_unlock(&fold_mutex);
  nsout1 = run_par.nfft/(2*run_par.nacc);      
  while(1)
  {
  	//pthread_mutex_lock(&data_mutex);
  	 semaphore_p(sem_fft3);
  	if(fft3Status[0]==1)
  	{
  	  //pointer = count*run_par.nfft/2;
  	  //fseek(fpin,pointer,SEEK_SET);
  	  //nread = fread(dataIn,1,run_par.nfft,fpin);
  	  printf("%d %d %d \n",dataIn[128*1024*1024],dataIn[0],run_par.nfft);
      fflush(stdout);
      gpufft();
  	  //if(nread!=run_par.nfft)
  	  //{
  		  //printf("the file might be at the end or there is some problem \n");
      //}
      /*
      for(i=0;i<run_par.nfft/2;i++)
      {
        //if(i==0) printf("the value of dataOut is %f %f \n",dataOut[0],dataOut[20]);
        dump+=dataOut[i];
      
        if((i+1)%run_par.nacc==0) 
        {
          tempOut[(i+1)/run_par.nacc-1] = (float)(dump/(long double)run_par.nacc);
          //printf("dump %Lf \n",dump); 
          dump = 0.0;
        } 
       
      }
      */
      
      fwrite(dataOut,sizeof(float),run_par.nfft/(2*run_par.nacc),fpout);
    
      totalSamples += run_par.nfft/2;
      printf("the number of samples = %ld and pointer = %ld  %d \n",totalSamples,pointer,dataIn[2]);
      fflush(stdout);
      count++;
      fft3Status[0]=0;
      //if(adc.fold == 1)
      {
        //pthread_mutex_lock(&fold_mutex);
        //check = foldVal;
        //pthread_mutex_unlock(&fold_mutex);
        check=1;
        /*
        while(check != 0)
        {
          pthread_mutex_lock(&fold_mutex);
          check = foldVal;
          pthread_mutex_unlock(&fold_mutex);
         
          pthread_mutex_lock(&foldStop_mutex);
          flag = foldStop;
          pthread_mutex_unlock(&foldStop_mutex);
         
          if(flag==0)
          {
            freeBuffers();
            printf("iam here .%d................ \n",foldVal);
            fflush(stdout);
            return(0);
          }  
        }
        */
        memcpy(timeSeries,tempOut,sizeof(float)*run_par.nfft/(2*run_par.nacc));
        pthread_mutex_lock(&fold_mutex);
        foldVal=1;
        
        pthread_mutex_unlock(&fold_mutex);
      }
      //*/
    }
    //pthread_mutex_unlock(&data_mutex);
    semaphore_v(sem_fft3);
    pthread_mutex_lock(&foldStop_mutex);
    flag = foldStop;
    pthread_mutex_unlock(&foldStop_mutex);
         
    if(flag==0) 
    {
      freeBuffers();
      return(0);
    }
  }
   
  
  freeBuffers(); 
  fclose(fpout);
  //fclose(fpin);
  //fclose(fpar);
}

/*
long double mjdnow()
{
  int dd, mm, yy;
  double a, b, c, e, f, jd;
  long double hr,mi,sec;
  
  time_t now;
  struct tm *d;
  struct tm *ptr_ts;
    
  time(&now);
  d = localtime(&now);
  time_t raw_time;
  time ( &raw_time );
  
  ptr_ts = gmtime ( &raw_time );
  hr = (ptr_ts->tm_hour)/24.0;
  mi = (ptr_ts->tm_min+1)/1440.0;
	
    yy = d->tm_year+1900;
    mm = d->tm_mon+1;
    dd = d->tm_mday;

	if (mm==1){
		yy -= 1;
		mm += 12;
	} else if(mm==2){
		yy -= 1;
		mm += 12;
	}

	a = yy/100.0; 
	b = a/4.0; 
	c = 2.0-a+b;
	e = 365.25*(yy+4716.0); 
	f = 30.6001*(mm+1.0);
	jd = c+dd+e+f-1525.0;
	int count=0;
	if (yy%4==0){
		
		count=1;
	} else if (yy%400==0){
		count=1;
		
	} 
	 
	if (count==1 && mm==14 && dd==29){
		jd -=1;
	}
	if (count!=1 && mm==3){
		jd -=1;
	}
	
  printf("%1.12Lf",(int)jd+hr+mi-2400000.0);
  return ((long double)((int)jd+hr+mi-2400000.0));
			
}	
*/
