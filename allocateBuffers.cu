#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
//#include<fftw3.h>
#include"struct.cuh"
#include"global.cuh"

void allocateBuffers(void)
{
   
	int i;
  printf("Allocating Buffers %d\n",run_par.nfft);
 // dataIn = (char *) malloc(sizeof(char)*run_par.nfft*2);
  printf("Allocated dataIn \n");
  dataOut = (float *) malloc(sizeof(float)*run_par.nfft/2);
  //temp = (float *) malloc(sizeof(float2)*(run_par.nfft));
  //chirp= (float2 *) malloc(sizeof(float2)*(run_par.nfft));
  checkCudaErrors(hipMalloc((void **)&tempIn, sizeof(char)*2*run_par.nfft));  
  //checkCudaErrors(hipMalloc((void **)&gpuChirp, sizeof(float2)*(run_par.nfft)));
  checkCudaErrors(hipMalloc((void **)&signal, sizeof(float2)*run_par.nfft));
  //checkCudaErrors(hipMalloc((void **)&t_signal, sizeof(float2)*(run_par.nfft/2+1)));
  //checkCudaErrors(hipMalloc((void **)&f_signal, sizeof(float2)*(run_par.nfft)));
  checkCudaErrors(hipMalloc((void **)&dev_nfft, sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&tmp, sizeof(float)*run_par.nfft));
  checkCudaErrors(hipMalloc((void **)&temp, sizeof(float)*run_par.nfft/run_par.nacc));
  
  checkCudaErrors(hipfftPlan1d(&planf, run_par.nfft, HIPFFT_C2C, 1));
  //checkCudaErrors(hipfftPlan1d(&planb, run_par.nfft, HIPFFT_C2R, 1));
  printf("The value of the nfft is %d \n",run_par.nfft); 
  //run_par.sideband = -1;
  //getchirp(chirp,run_par.nfft,-1,334.5,run_par.bw,run_par.dm,0.0);
  //printf("Copying the chirp funtion");
  //hipMemcpy(gpuChirp, chirp, (run_par.nfft)*sizeof(float2),hipMemcpyHostToDevice);
  hipMemcpy(dev_nfft, &run_par.nfft,sizeof(int),hipMemcpyHostToDevice);   
  fflush(stdout);  
  
}
