#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "struct.cuh"
#include "global.cuh"

#define TWOPI 6.2831853071796
#define DFFAC 2.41e-10 

 
__global__ void complexMul( float2 *a, int *N,double dm,double fsky, int sideband,double bw) 
{
  //int tid = blockIdx.x;    // this thread handles the data at its thread id
  //int tid = threadIdx.x;
  float2 t;
  float2 c;
  double f,s,r,taper;
  float2 at;
  
  s = TWOPI*dm/(DFFAC);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
  at = a[tid];
  
  f = tid*bw/(*N/2);
  if(f > bw) 
  {
    f -= bw; 
    f = bw -f;
  }
  if(tid<=(*N)/2) r = -1*f*f*s/((fsky+sideband*f)*fsky*fsky);  
  else r = f*f*s/((fsky+sideband*f)*fsky*fsky);
    
  if (f > 0.5*bw) taper = 1.0/sqrt(1.0 + pow((f/(0.94*bw)),80));
  else  taper = 1.0/sqrt(1.0 + pow(((bw-f)/(0.84*bw)),80));
  c.x = (float)( cos(r) * taper );
  c.y = ( -1.0* (float)( sin(r) * taper));
    
  if (tid < N[0])
  {
    t.x = ((at.x*c.x)-(at.y*c.y))/10000;
    t.y = ((at.x*c.y)+(at.y*c.x))/10000;
    a[tid].x = t.x;
    a[tid].y = t.y;
  }
}

__global__ void typecaste( float2 *a, char *b, int *N)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //if(tid==10) printf("the value of the dataIn %d \n",b[tid]);
  if (tid < *N)
  {
    a[tid].x = ((float)b[2*tid]+(float)b[2*tid+1])/2; 
    a[tid].y = 0.0;
  }
}

__global__ void scale( float2 *a,float *b,int *N)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //if(tid==10) printf("the value of the dataIn %d \n",b[tid]);
  if (tid < *N)
  {
    b[tid] = ((a[tid].x*a[tid].x) + (a[tid].y*a[tid].y))/(134217728.0); 
  }
}
 
__global__ void integrate( float *a,float *b,int nacc, int nfft)
{
  float t=0.0;
  int i;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //if(tid==10) printf("the value of the dataIn %d \n",b[tid]);
  for(i=0;i<nacc;i++)
  {
    t+=b[tid*nacc+i];
  }
  
  t = t/nacc;
  
  a[tid] = t;
}
 
void gpufft()
{
  
   
  checkCudaErrors(hipMemcpy(tempIn, dataIn, 2*run_par.nfft*sizeof(char),hipMemcpyHostToDevice));
  typecaste<<<run_par.nfft/1024,1024>>>(signal,tempIn,dev_nfft);
  checkCudaErrors(hipfftExecC2C(planf, signal, signal, HIPFFT_FORWARD));
  
  complexMul<<<run_par.nfft/1024,1024>>>( signal, dev_nfft,run_par.dm,334.5,-1,16 );
  checkCudaErrors(hipfftExecC2C(planf, signal,signal, HIPFFT_BACKWARD)); 
  scale<<<run_par.nfft/1024,1024>>>(signal,tmp,dev_nfft);
  integrate<<<(run_par.nfft/run_par.nacc)/1024,1024>>>( temp,tmp, run_par.nacc, run_par.nfft);
  checkCudaErrors(hipMemcpy(dataOut,&temp[run_par.nfft/(4*run_par.nacc)] , 
                            sizeof(float)*(run_par.nfft/(2*run_par.nacc)),hipMemcpyDeviceToHost));
   
  return;
}
 
 
 
